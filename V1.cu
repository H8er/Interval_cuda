
#include <hip/hip_runtime.h>
#include <iostream>

using namespace std;

#define TYPE double
typedef TYPE T;
// int threads = 1;
// int blocks = 1;

__constant__ double dev_box[4];
__constant__ int dev_threads[1];
__constant__ int dev_blocks[1];
__constant__ int dev_n_of_ints[1];

template<class T>

class interval_gpu
{
    public:
        __device__ __host__ interval_gpu();
        __device__ __host__ interval_gpu(T const &v);
        __device__ __host__ interval_gpu(T const &l, T const &u);
        __device__ __host__ T const &lower() const;
        __device__ __host__ T const &upper() const;
        static __device__ __host__ interval_gpu empty();
friend ostream& operator<<(ostream& os, const interval_gpu<T> &x){
os<<"["<<x.lower()<<":"<<x.upper()<<"]";return os;
}
    private: T low; T up;
};
// Constructors
template<class T> inline __device__ __host__
interval_gpu<T>::interval_gpu(){}
template<class T> inline __device__ __host__
interval_gpu<T>::interval_gpu(T const &v) :
    low(v), up(v){}
template<class T> inline __device__ __host__
interval_gpu<T>::interval_gpu(T const &l, T const &u) :
    low(l), up(u){}

template<class T> inline __device__ __host__
T const &interval_gpu<T>::lower() const
{return low;}

template<class T> inline __device__ __host__
T const &interval_gpu<T>::upper() const
{return up;}
//OVERLOAD OVERLOAD OVERLOAD OVERLOAD OVERLOAD OVERLOAD OVERLOAD OVERLOAD OVERLOAD
template<class T> inline __host__ __device__
interval_gpu<T> operator+(interval_gpu<T> const &x, interval_gpu<T> const &y)
{
  return interval_gpu<T>(x.lower() + y.lower(), x.upper() + y.upper());
}
template<class T> inline __host__ __device__
interval_gpu<T> operator-(interval_gpu<T> const &x, interval_gpu<T> const &y)
{return interval_gpu<T>(x.lower() - y.upper(), x.upper() - y.lower());}
template<class T> inline __host__ __device__
interval_gpu<T> operator*(interval_gpu<T> const &x, interval_gpu<T> const &y)
{return interval_gpu<T>(min(min(x.lower()*y.lower(),x.lower()*y.upper()),
          min(x.upper()*y.lower(),x.upper()*y.upper())),
        max(max(x.lower()*y.lower(),x.lower()*y.upper()),
            max(x.upper()*y.lower(),x.upper()*y.upper())));}
template<class T> inline __host__ __device__
interval_gpu<T> operator/(interval_gpu<T> const &x, interval_gpu<T> const &y)
{return interval_gpu<T>(min(min(x.lower()/y.lower(),x.lower()/y.upper()),
          min(x.upper()/y.lower(),x.upper()/y.upper())),
        max(max(x.lower()/y.lower(),x.lower()/y.upper()),
            max(x.upper()/y.lower(),x.upper()/y.upper())));}


template<class T> inline __device__
int g1(interval_gpu<T> *x){
interval_gpu<T> lmax(12);
// (x[0]*x[0] + x2*x[1] - lmax*lmax)
interval_gpu<T> f(x[0]*x[0] + x[1]*x[1] - lmax*lmax);
return int(bool(f.upper() < 0) + bool(f.lower() < 0));
}
template<class T> inline __device__
int g2(interval_gpu<T> *x){
interval_gpu<T> l(8);
interval_gpu<T> f(l*l - x[0]*x[0] - x[1]*x[1]);
return int(bool(f.upper() < 0) + bool(f.lower() < 0));
}

template<class T> inline __device__
int g3(interval_gpu<T> *x){
interval_gpu<T> lmax(12);
interval_gpu<T> l0(5);
interval_gpu<T> f((x[0]-l0)*(x[0]-l0) + x[1]*x[1] - lmax*lmax);
return int(bool(f.upper() < 0) + bool(f.lower() < 0));
}
template<class T> inline __device__
int g4(interval_gpu<T> *x){
interval_gpu<T> l(8);
interval_gpu<T> l0(5);
interval_gpu<T> f(l*l  - (x[0]-l0)*(x[0]-l0) - x[1]*x[1]);
return int(bool(f.upper() < 0) + bool(f.lower() < 0));
}

template<class T> inline __host__ __device__
int gg(interval_gpu<T>* xi, int n){
interval_gpu<T> lmax(12);
interval_gpu<T> f(xi[0]*xi[0] + xi[1]*xi[1] - lmax*lmax);
return int(bool(f.upper() < 0) + bool(f.lower() < 0));
}


template<class T>
__global__ void first_grid(int* res){
  interval_gpu<T>* x = new interval_gpu<T>[dev_n_of_ints[0]];
  x[0] = interval_gpu<T>(dev_box[0] +  (threadIdx.x) * (dev_box[1] - dev_box[0])/64,
                         dev_box[0] +(1+threadIdx.x) * (dev_box[1] - dev_box[0])/64);
  x[1] = interval_gpu<T>(dev_box[2] +   (blockIdx.x) * (dev_box[3] - dev_box[2])/32,
                         dev_box[2] + (1+blockIdx.x) * (dev_box[3] - dev_box[2])/32);

  int (*func_pointers[4])(interval_gpu<T>*) = {&g1,&g2,&g3,&g4};
  res[(blockIdx.x*blockDim.x + threadIdx.x)*16] = 1;

  for(int i = 0; i < 4; i++){
    res[(blockIdx.x*blockDim.x + threadIdx.x)*16] *= (*func_pointers[i])(x);
  }

  res[(blockIdx.x*blockDim.x + threadIdx.x)*16] = 9;
}

template<class T>
__global__ void second_grid(int* res){
  // if((res[blockIdx.x*blockDim.x] > 0)and(res[blockIdx.x*blockDim.x] < 8)){
    interval_gpu<T>* x = new interval_gpu<T>[dev_n_of_ints[0]];
    x[0] = interval_gpu<T>(dev_box[0] +  (threadIdx.x) * (dev_box[1] - dev_box[0])/dev_threads[0],
                           dev_box[0] +(1+threadIdx.x) * (dev_box[1] - dev_box[0])/dev_threads[0]);
    x[1] = interval_gpu<T>(dev_box[2] +   (blockIdx.x) * (dev_box[3] - dev_box[2])/dev_blocks[0],
                           dev_box[2] + (1+blockIdx.x) * (dev_box[3] - dev_box[2])/dev_blocks[0]);
                           int (*func_pointers[4])(interval_gpu<T>*) = {&g1,&g2,&g3,&g4};
    res[blockIdx.x*blockDim.x + threadIdx.x] = 1;
    for(int i = 0; i < 4; i++){
      res[blockIdx.x*blockDim.x + threadIdx.x] *= (*func_pointers[i])(x);
    }
  // }

  if(res[(blockIdx.x*blockDim.x + threadIdx.x)] == 9){
    res[blockIdx.x*blockDim.x + threadIdx.x] = 3;
  }
}

int main(){
    int n_of_ints = 2;

    double host_box[4] = {-15.0,15.0,0.0,15.0};

    int ithreads = 256;
    int iblocks = 128;
    int * res;
    cout<<fixed;
    cout.precision(4);
    hipMallocManaged(&res, sizeof(int)*iblocks*ithreads);
    hipMemcpyToSymbol(HIP_SYMBOL(dev_n_of_ints), &n_of_ints, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(dev_threads), &ithreads, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(dev_blocks), &iblocks, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(dev_box), &host_box, sizeof(double)*4);

    first_grid<T><<<32, 64>>>(res);
    hipDeviceSynchronize();
      for(int i = 0; i < 32; i++){
        for(int j = 0; j < 64; j++){
        cout<<res[(i*ithreads+j)*16]<<"\n";
        // if(int(res[(i*64+j)]) > 0){
        //         interval_gpu<T> x1(host_box[0] + (j) * (host_box[1] - host_box[0])/64 ,host_box[0]+(1+j) * (host_box[1] - host_box[0])/64);
        //         interval_gpu<T> x2(host_box[2] + (i) * (host_box[3] - host_box[2])/32 ,host_box[2]+(1+i) * (host_box[3] - host_box[2])/32);
        //         cout<<x1<<":"<<x2<<"\n";
        // }
      }
  }

    // for(int i = 0; i < 10; i++){
    //   for(int j = 0; j < 32; j++){
    //     if(int(res[i*32+j]) > 0){
    //       interval_gpu<T> x1(host_box[0] + (j) * (host_box[1] - host_box[0])/32 ,host_box[0]+(1+j) * (host_box[1] - host_box[0])/32);
    //       interval_gpu<T> x2(host_box[2] + (i) * (host_box[3] - host_box[2])/10 ,host_box[2]+(1+i) * (host_box[3] - host_box[2])/10);
    //       cout<<x1<<":"<<x2<<"\n";
    //     }
    //   }
    // }
  //   cout<<"# "<<iblocks<<" x "<<ithreads<<"\n";
    second_grid<T><<<iblocks, ithreads>>>(res);
    hipDeviceSynchronize();
    for(int i = 0; i < iblocks; i++){
      for(int j = 0; j < ithreads; j++){
        cout<<res[i*ithreads+j]<<"\n";
        // if(int(res[i*ithreads+j]) > 0){
        //   interval_gpu<T> x1(host_box[0] + (j) * (host_box[1] - host_box[0])/ithreads ,host_box[0]+(1+j) * (host_box[1] - host_box[0])/ithreads );
        //   interval_gpu<T> x2(host_box[2] + (i) * (host_box[3] - host_box[2])/iblocks ,host_box[2]+(1+i) * (host_box[3] - host_box[2])/iblocks);
        //   cout<<x1<<":"<<x2<<"\n";
        // }
      }
  }

    hipFree(res);
    hipFree(dev_blocks);
    hipFree(dev_threads);

    return 0;
}
